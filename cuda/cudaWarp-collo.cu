#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, Hiroshi Yamamoto. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include "cudaWarp.h"
#include "cudaFilterMode.cuh"


// cudaCollo
template<typename T, typename S>
__global__ void cudaCollo( T* input, S* output, st_COLLO_param collo_prm )
{
	const int2 uv_out = make_int2(blockDim.x * blockIdx.x + threadIdx.x,
				               blockDim.y * blockIdx.y + threadIdx.y);
						   
	if( uv_out.x >= collo_prm.oW || uv_out.y >= collo_prm.oH )
		return;
	
	const int iW  = collo_prm.iW;
	const int iH = collo_prm.iH;
	const int oW  = collo_prm.oW;
	const int oH = collo_prm.oH;
	const float iW_f  = iW;
	const float iH_f = iH;
	const float oW_f  = oW;
	const float oH_f = oH;

	const float fov = collo_prm.v_fov_half_tan;
	const float k = collo_prm.lens_radius_scale;
	
	// convert to cartesian coordinates
	const float cx = ((uv_out.x / oW_f) - 0.5f) * 2.0f * collo_prm.oAspect;	
	const float cy = ((uv_out.y / oH_f) - 0.5f) * 2.0f;

	// 2D -> 3D.
	// right-handed system. x: right(->), y: down(|v), z: far(X).
	float3 po = {
		.x = cx * fov,
		.y = cy * fov,
		.z = 1.0f,
	};

	// roll(inner).
	float3 p_roll = rotZ(po, collo_prm.rot.roll);

	// pan, tilt.
	float3 p_rot = p_roll;
	if (collo_prm.absolute) {
		p_rot = rotX(p_rot, -collo_prm.target_y * M_PI / 4.0f);	// X. tilt.
		p_rot = rotY(p_rot, collo_prm.target_x * M_PI / 4.0f);	// Y. pan.
		p_rot = rotZ(p_rot, collo_prm.rot.z);	// Z. outer roll.
	} else {
		p_rot = rotX(p_rot, collo_prm.rot.x);	// X. tilt.
		p_rot = rotY(p_rot, collo_prm.rot.y);	// Y. pan.
		p_rot = rotZ(p_rot, collo_prm.rot.z);	// Z. outer roll.
	}

	// normalized sphere. r = 1.0.
	float3 p_sph = normalize(p_rot);

	// XYZ -> theta_x, theta_z.
	float theta_x = atan2f(p_sph.y, p_sph.x);
	float theta_z = acosf(p_sph.z);

	// 3D -> 2D.
	float r = f_lens_radius(theta_z, k);
	float tx = r * __cosf(theta_x);
	float ty = r * __sinf(theta_x);

	// -> XY(input). with adjustment of lens center.
	float u = ((tx * 0.5f / collo_prm.iAspect) + 0.5f) * iW_f;
	float v = ((ty * 0.5f) + 0.5f) * iH_f;
	u += collo_prm.xcenter;
	v += collo_prm.ycenter;

	if( u < 0.0f ) u = 0.0f;
	if( v < 0.0f ) v = 0.0f;

	if( u > iW_f - 1.0f ) u = iW_f - 1.0f;
	if( v > iH_f - 1.0f ) v = iH_f - 1.0f;
	
	// sampling pixel.
	float2 scale = { 1.0f, 1.0f };
	float max_value = 255.0f;
	switch (collo_prm.filter_mode) {
	case FILTER_LINEAR:	// Bi-linear. 3x3 filter.
		output[uv_out.y * oW + uv_out.x] = cast_vec<S>(cudaFilterPixel<FILTER_LINEAR>(input, u, v, iW, iH, oW, oH, scale, max_value));
		break;
	case FILTER_CUBIC:	// Bi-cubic. 5x5 filter.
		output[uv_out.y * oW + uv_out.x] = cast_vec<S>(cudaFilterPixel<FILTER_CUBIC>(input, u, v, iW, iH, oW, oH, scale, max_value));
		break;
	case FILTER_SPLINE36:	// Spline36. 7x7 filter.
		output[uv_out.y * oW + uv_out.x] = cast_vec<S>(cudaFilterPixel<FILTER_SPLINE36>(input, u, v, iW, iH, oW, oH, scale, max_value));
		break;
	case FILTER_LANCZOS4:	// slowest. Lanczos4. 9x9 filter.
		output[uv_out.y * oW + uv_out.x] = cast_vec<S>(cudaFilterPixel<FILTER_LANCZOS4>(input, u, v, iW, iH, oW, oH, scale, max_value));
		break;
	case FILTER_POINT:	// fastest. nearest.
	default:
		output[uv_out.y * oW + uv_out.x] = cast_vec<S>(cudaFilterPixel<FILTER_POINT>(input, u, v, iW, iH, oW, oH, scale, max_value));
	}
}


// cudaWarpCollo
template<typename T, typename S> inline hipError_t cudaWarpCollo__( T* input, S* output, st_COLLO_param collo_prm )
{
	if( !input || !output )
		return hipErrorInvalidDevicePointer;

	if( collo_prm.iW == 0 || collo_prm.iH == 0 || collo_prm.oW == 0 || collo_prm.oH == 0 )
		return hipErrorInvalidValue;

	// launch kernel
	const dim3 blockDim(8, 8);
	const dim3 gridDim(iDivUp(collo_prm.oW,blockDim.x), iDivUp(collo_prm.oH,blockDim.y));

	cudaCollo<T, S><<<gridDim, blockDim>>>(input, output, collo_prm);

	return CUDA(hipGetLastError());
}
#define FUNC_CUDA_WARP_COLLO(T, S) \
hipError_t cudaWarpCollo( T* input, S* output, st_COLLO_param collo_prm ) { return cudaWarpCollo__<T, S>( input, output, collo_prm ); }

// cudaWarpCollo (uint8 grayscale)
FUNC_CUDA_WARP_COLLO(uint8_t, uint8_t);
FUNC_CUDA_WARP_COLLO(float, uint8_t);
FUNC_CUDA_WARP_COLLO(uchar3, uint8_t);
FUNC_CUDA_WARP_COLLO(uchar4, uint8_t);
FUNC_CUDA_WARP_COLLO(float3, uint8_t);
FUNC_CUDA_WARP_COLLO(float4, uint8_t);

// cudaWarpCollo (float grayscale)
FUNC_CUDA_WARP_COLLO(uint8_t, float);
FUNC_CUDA_WARP_COLLO(float, float);
FUNC_CUDA_WARP_COLLO(uchar3, float);
FUNC_CUDA_WARP_COLLO(uchar4, float);
FUNC_CUDA_WARP_COLLO(float3, float);
FUNC_CUDA_WARP_COLLO(float4, float);

// cudaWarpCollo (uchar3)
FUNC_CUDA_WARP_COLLO(uint8_t, uchar3);
FUNC_CUDA_WARP_COLLO(float, uchar3);
FUNC_CUDA_WARP_COLLO(uchar3, uchar3);
FUNC_CUDA_WARP_COLLO(uchar4, uchar3);
FUNC_CUDA_WARP_COLLO(float3, uchar3);
FUNC_CUDA_WARP_COLLO(float4, uchar3);

// cudaWarpCollo (uchar4)
FUNC_CUDA_WARP_COLLO(uint8_t, uchar4);
FUNC_CUDA_WARP_COLLO(float, uchar4);
FUNC_CUDA_WARP_COLLO(uchar3, uchar4);
FUNC_CUDA_WARP_COLLO(uchar4, uchar4);
FUNC_CUDA_WARP_COLLO(float3, uchar4);
FUNC_CUDA_WARP_COLLO(float4, uchar4);

// cudaWarpCollo (float3)
FUNC_CUDA_WARP_COLLO(uint8_t, float3);
FUNC_CUDA_WARP_COLLO(float, float3);
FUNC_CUDA_WARP_COLLO(uchar3, float3);
FUNC_CUDA_WARP_COLLO(uchar4, float3);
FUNC_CUDA_WARP_COLLO(float3, float3);
FUNC_CUDA_WARP_COLLO(float4, float3);

// cudaWarpCollo (float4)
FUNC_CUDA_WARP_COLLO(uint8_t, float4);
FUNC_CUDA_WARP_COLLO(float, float4);
FUNC_CUDA_WARP_COLLO(uchar3, float4);
FUNC_CUDA_WARP_COLLO(uchar4, float4);
FUNC_CUDA_WARP_COLLO(float3, float4);
FUNC_CUDA_WARP_COLLO(float4, float4);

#undef FUNC_CUDA_WARP_COLLO
