#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2017, NVIDIA CORPORATION. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include "cudaResize.h"
#include "cudaFilterMode.cuh"



// gpuResize. nearest.
template<typename T>
__global__ void gpuResize_nearest( float2 scale, T* input, int iWidth, int iHeight, T* output, int oWidth, int oHeight )
{
	const int dst_x = blockIdx.x * blockDim.x + threadIdx.x;
	const int dst_y = blockIdx.y * blockDim.y + threadIdx.y;

	if (dst_x < oWidth && dst_y < oHeight)
	{
		const T px = cudaFilterPixel<FILTER_POINT>(input, dst_x, dst_y, iWidth, iHeight, oWidth, oHeight, scale);
		output[dst_y * oWidth + dst_x] = px;
	}
}

// gpuResize. linear.
template <typename T>
__global__ void gpuResize_linear( float2 scale, T* input, int iWidth, int iHeight, T* output, int oWidth, int oHeight, float max_value )
{
	const int dst_x = blockDim.x * blockIdx.x + threadIdx.x;
	const int dst_y = blockDim.y * blockIdx.y + threadIdx.y;

	if (dst_x < oWidth && dst_y < oHeight)
	{
		T out = cudaFilterPixel<FILTER_LINEAR>(input, dst_x, dst_y, iWidth, iHeight, oWidth, oHeight, scale, max_value);
		output[dst_y * oWidth + dst_x] = out;
	}
}

// gpuResize. cubic.
template <typename T>
__global__ void gpuResize_cubic( float2 scale, T* input, int iWidth, int iHeight, T* output, int oWidth, int oHeight, float max_value )
{
	const int dst_x = blockDim.x * blockIdx.x + threadIdx.x;
	const int dst_y = blockDim.y * blockIdx.y + threadIdx.y;

	if (dst_x < oWidth && dst_y < oHeight)
	{
		T out = cudaFilterPixel<FILTER_CUBIC>(input, dst_x, dst_y, iWidth, iHeight, oWidth, oHeight, scale, max_value);
		output[dst_y * oWidth + dst_x] = out;
	}
}

// gpuResize. area.
template <typename T>
__global__ void gpuResize_area( float2 scale, T* input, int iWidth, int iHeight, T* output, int oWidth, int oHeight, float max_value )
{
	const int dst_x = blockIdx.x * blockDim.x + threadIdx.x;
	const int dst_y = blockIdx.y * blockDim.y + threadIdx.y;

	if (dst_x < oWidth && dst_y < oHeight)
	{
		T out = cudaFilterPixel<FILTER_AREA>(input, dst_x, dst_y, iWidth, iHeight, oWidth, oHeight, scale, max_value);
		output[dst_y * oWidth + dst_x] = out;
	}
}

// gpuResize. lanczos4.
template <typename T>
__global__ void gpuResize_lanczos4( float2 scale, T* input, int iWidth, int iHeight, T* output, int oWidth, int oHeight, float max_value )
{
	const int dst_x = blockDim.x * blockIdx.x + threadIdx.x;
	const int dst_y = blockDim.y * blockIdx.y + threadIdx.y;

	if (dst_x < oWidth && dst_y < oHeight)
	{
		T out = cudaFilterPixel<FILTER_LANCZOS4>(input, dst_x, dst_y, iWidth, iHeight, oWidth, oHeight, scale, max_value);
		output[dst_y * oWidth + dst_x] = out;
	}
}

// gpuResize. spline36.
template <typename T>
__global__ void gpuResize_spline36( float2 scale, T* input, int iWidth, int iHeight, T* output, int oWidth, int oHeight, float max_value )
{
	const int dst_x = blockDim.x * blockIdx.x + threadIdx.x;
	const int dst_y = blockDim.y * blockIdx.y + threadIdx.y;

	if (dst_x < oWidth && dst_y < oHeight)
	{
		T out = cudaFilterPixel<FILTER_SPLINE36>(input, dst_x, dst_y, iWidth, iHeight, oWidth, oHeight, scale, max_value);
		output[dst_y * oWidth + dst_x] = out;
	}
}

// launchResize
template<typename T>
static hipError_t launchResize( T* input, size_t inputWidth, size_t inputHeight,
				             T* output, size_t outputWidth, size_t outputHeight, int mode, float max_value )
{
	if( !input || !output )
		return hipErrorInvalidDevicePointer;

	if( inputWidth == 0 || outputWidth == 0 || inputHeight == 0 || outputHeight == 0 )
		return hipErrorInvalidValue;

	const float2 scale = make_float2( float(inputWidth) / float(outputWidth),
							     float(inputHeight) / float(outputHeight) );

	// launch kernel
	const dim3 blockDim(32, 8);
	const dim3 gridDim(iDivUp(outputWidth,blockDim.x), iDivUp(outputHeight,blockDim.y));

	if (mode == static_cast<int>(InterpolationFlags::INTER_LINEAR)) {
		if (inputWidth == outputWidth && inputHeight == outputHeight) {
			gpuResize_nearest<T><<<gridDim, blockDim>>>(scale, input, inputWidth, inputHeight, output, outputWidth, outputHeight);
		} else {
			gpuResize_linear<T><<<gridDim, blockDim>>>(scale, input, inputWidth, inputHeight, output, outputWidth, outputHeight, max_value);
		}

	} else if (mode == static_cast<int>(InterpolationFlags::INTER_CUBIC)) {
		if (inputWidth == outputWidth && inputHeight == outputHeight) {
			gpuResize_nearest<T><<<gridDim, blockDim>>>(scale, input, inputWidth, inputHeight, output, outputWidth, outputHeight);
		} else {
			gpuResize_cubic<T><<<gridDim, blockDim>>>(scale, input, inputWidth, inputHeight, output, outputWidth, outputHeight, max_value);
		}

	} else if (mode == static_cast<int>(InterpolationFlags::INTER_AREA)) {
		if (inputWidth == outputWidth && inputHeight == outputHeight) {
			gpuResize_nearest<T><<<gridDim, blockDim>>>(scale, input, inputWidth, inputHeight, output, outputWidth, outputHeight);
		} else if (inputWidth < outputWidth || inputHeight < outputHeight) {
			gpuResize_linear<T><<<gridDim, blockDim>>>(scale, input, inputWidth, inputHeight, output, outputWidth, outputHeight, max_value);
		} else {
			gpuResize_area<T><<<gridDim, blockDim>>>(scale, input, inputWidth, inputHeight, output, outputWidth, outputHeight, max_value);
		}

	} else if (mode == static_cast<int>(InterpolationFlags::INTER_LANCZOS4)) {
		if (inputWidth == outputWidth && inputHeight == outputHeight) {
			gpuResize_nearest<T><<<gridDim, blockDim>>>(scale, input, inputWidth, inputHeight, output, outputWidth, outputHeight);
		} else {
			gpuResize_lanczos4<T><<<gridDim, blockDim>>>(scale, input, inputWidth, inputHeight, output, outputWidth, outputHeight, max_value);
		}

	} else if (mode == static_cast<int>(InterpolationFlags::INTER_SPLINE36)) {
		if (inputWidth == outputWidth && inputHeight == outputHeight) {
			gpuResize_nearest<T><<<gridDim, blockDim>>>(scale, input, inputWidth, inputHeight, output, outputWidth, outputHeight);
		} else {
			gpuResize_spline36<T><<<gridDim, blockDim>>>(scale, input, inputWidth, inputHeight, output, outputWidth, outputHeight, max_value);
		}

	} else {
		gpuResize_nearest<T><<<gridDim, blockDim>>>(scale, input, inputWidth, inputHeight, output, outputWidth, outputHeight);
	}

	return CUDA(hipGetLastError());
}

// cudaResize (uint8 grayscale)
hipError_t cudaResize( uint8_t* input, size_t inputWidth, size_t inputHeight, uint8_t* output, size_t outputWidth, size_t outputHeight, int mode, float max_value )
{
	return launchResize<uint8_t>(input, inputWidth, inputHeight, output, outputWidth, outputHeight, mode, max_value);
}

// cudaResize (float grayscale)
hipError_t cudaResize( float* input, size_t inputWidth, size_t inputHeight, float* output, size_t outputWidth, size_t outputHeight, int mode, float max_value )
{
	return launchResize<float>(input, inputWidth, inputHeight, output, outputWidth, outputHeight, mode, max_value);
}

// cudaResize (uchar3)
hipError_t cudaResize( uchar3* input, size_t inputWidth, size_t inputHeight, uchar3* output, size_t outputWidth, size_t outputHeight, int mode, float max_value )
{
	return launchResize<uchar3>(input, inputWidth, inputHeight, output, outputWidth, outputHeight, mode, max_value);
}

// cudaResize (uchar4)
hipError_t cudaResize( uchar4* input, size_t inputWidth, size_t inputHeight, uchar4* output, size_t outputWidth, size_t outputHeight, int mode, float max_value )
{
	return launchResize<uchar4>(input, inputWidth, inputHeight, output, outputWidth, outputHeight, mode, max_value);
}

// cudaResize (float3)
hipError_t cudaResize( float3* input, size_t inputWidth, size_t inputHeight, float3* output, size_t outputWidth, size_t outputHeight, int mode, float max_value )
{
	return launchResize<float3>(input, inputWidth, inputHeight, output, outputWidth, outputHeight, mode, max_value);
}

// cudaResize (float4)
hipError_t cudaResize( float4* input, size_t inputWidth, size_t inputHeight, float4* output, size_t outputWidth, size_t outputHeight, int mode, float max_value )
{
	return launchResize<float4>(input, inputWidth, inputHeight, output, outputWidth, outputHeight, mode, max_value);
}

//-----------------------------------------------------------------------------------
hipError_t cudaResize( void* input,  size_t inputWidth,  size_t inputHeight,
				    void* output, size_t outputWidth, size_t outputHeight, imageFormat format, int mode )
{
	if( format == IMAGE_RGB8 || format == IMAGE_BGR8 )
		return cudaResize((uchar3*)input, inputWidth, inputHeight, (uchar3*)output, outputWidth, outputHeight, mode);
	else if( format == IMAGE_RGBA8 || format == IMAGE_BGRA8 )
		return cudaResize((uchar4*)input, inputWidth, inputHeight, (uchar4*)output, outputWidth, outputHeight, mode);
	else if( format == IMAGE_RGB32F || format == IMAGE_BGR32F )
		return cudaResize((float3*)input, inputWidth, inputHeight, (float3*)output, outputWidth, outputHeight, mode);
	else if( format == IMAGE_RGBA32F || format == IMAGE_BGRA32F )
		return cudaResize((float4*)input, inputWidth, inputHeight, (float4*)output, outputWidth, outputHeight, mode);
	else if( format == IMAGE_GRAY8 )
		return cudaResize((uint8_t*)input, inputWidth, inputHeight, (uint8_t*)output, outputWidth, outputHeight, mode);
	else if( format == IMAGE_GRAY32F )
		return cudaResize((float*)input, inputWidth, inputHeight, (float*)output, outputWidth, outputHeight, mode);

	LogError(LOG_CUDA "cudaResize() -- invalid image format '%s'\n", imageFormatToStr(format));
	LogError(LOG_CUDA "                supported formats are:\n");
	LogError(LOG_CUDA "                    * gray8\n");
	LogError(LOG_CUDA "                    * gray32f\n");
	LogError(LOG_CUDA "                    * rgb8, bgr8\n");
	LogError(LOG_CUDA "                    * rgba8, bgra8\n");
	LogError(LOG_CUDA "                    * rgb32f, bgr32f\n");
	LogError(LOG_CUDA "                    * rgba32f, bgra32f\n");

	return hipErrorInvalidValue;
}
